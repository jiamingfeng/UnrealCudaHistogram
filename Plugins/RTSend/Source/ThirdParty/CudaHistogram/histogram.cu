#include "hip/hip_runtime.h"
#include <d3d11.h>
#include <cuda_d3d11_interop.h>
#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include <hip/hip_runtime.h>
#include <sstream>
#include "cub/device/device_histogram.cuh"

extern "C"
std::string GenerateHistogram(ID3D11Texture2D* dxTexture, int width, int height, int* Histogram);

#define BIN_COUNT 256
#define HISTOGRAM_SIZE (BIN_COUNT * sizeof(unsigned int))

texture<uchar4, 2, hipReadModeElementType> colorTex;

////////////////////////////////////////////////////////////////////////////////
// GPU-specific definitions
////////////////////////////////////////////////////////////////////////////////
//Fast mul on G8x / G9x / G100
#define IMUL(a, b) __mul24(a, b)

//Machine warp size
//G80's warp size is 32 threads
#define WARP_LOG2SIZE 5

//Warps in thread block for histogram256Kernel()
#define WARP_N 6

//Corresponding thread block size in threads for histogram256Kernel()
#define THREAD_N (WARP_N << WARP_LOG2SIZE)

//Total histogram size (in counters) per thread block for histogram256Kernel()
#define BLOCK_MEMORY (WARP_N * BIN_COUNT)

//Thread block count for histogram256Kernel()
#define BLOCK_N 64

#define TAG_MASK 0x07FFFFFFU//0x070707FFU//


////////////////////////////////////////////////////////////////////////////////
// If threadPos == threadIdx.x, there are always  4-way bank conflicts,
// since each group of 16 threads (half-warp) accesses different bytes,
// but only within 4 shared memory banks. Having shuffled bits of threadIdx.x
// as in histogram64GPU(), each half-warp accesses different shared memory banks
// avoiding any bank conflicts at all.
// Refer to the supplied whitepaper for detailed explanations.
////////////////////////////////////////////////////////////////////////////////
__device__ inline void addData256(volatile unsigned int *s_WarpHist, unsigned int data, unsigned int threadTag)
{
	unsigned int count;

	do
	{
		count = s_WarpHist[data] & TAG_MASK;
		count = threadTag | (count + 1);
		s_WarpHist[data] = count;
	} while (s_WarpHist[data] != count);
}

////////////////////////////////////////////////////////////////////////////////
// Main histogram calculation kernel
////////////////////////////////////////////////////////////////////////////////
static __global__ void histogramTex256Kernel(int *d_Result, unsigned int width, unsigned int height, int dataN)
{
	//Current global thread index
	const int    globalTid = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	//Total number of threads in the compute grid
	const int   numThreads = IMUL(blockDim.x, gridDim.x);

	//Thread tag for addData256()
	//WARP_LOG2SIZE higher bits of counter values are tagged
	//by lower WARP_LOG2SIZE threadID bits
	const unsigned int threadTag = threadIdx.x << (32 - WARP_LOG2SIZE);

	//Shared memory storage for each warp
	volatile __shared__ unsigned int s_Hist[BLOCK_MEMORY];

	//Current warp shared memory base
	const int warpBase = (threadIdx.x >> WARP_LOG2SIZE) * BIN_COUNT;

	//Clear shared memory buffer for current thread block before processing
	for (int pos = threadIdx.x; pos < BLOCK_MEMORY; pos += blockDim.x)
		s_Hist[pos] = 0;

	//Cycle through the entire data set, update subhistograms for each warp
	__syncthreads();

	for (int pos = globalTid; pos < dataN; pos += numThreads)
	{
		// NOTE: check this... Not sure this is what needs to be done
		int py = pos / width;
		int px = pos - (py * width);
		uchar4 data4 = tex2D(colorTex, px, py);

		addData256(s_Hist + warpBase, (data4.x), threadTag);
		addData256(s_Hist + warpBase, (data4.y), threadTag);
		addData256(s_Hist + warpBase, (data4.z), threadTag);
		addData256(s_Hist + warpBase, (data4.w), threadTag);
	}

	__syncthreads();

	//Merge per-warp histograms into per-block and write to global memory
	for (int pos = threadIdx.x; pos < BIN_COUNT; pos += blockDim.x)
	{
		unsigned int sum = 0;

		for (int base = 0; base < BLOCK_MEMORY; base += BIN_COUNT)
			sum += s_Hist[base + pos] & TAG_MASK;

		d_Result[blockIdx.x * BIN_COUNT + pos] = int(sum);
	}
}

///////////////////////////////////////////////////////////////////////////////
// Merge BLOCK_N subhistograms of BIN_COUNT bins into final histogram
///////////////////////////////////////////////////////////////////////////////
// gridDim.x   == BIN_COUNT
// blockDim.x  == BLOCK_N
// blockIdx.x  == bin counter processed by current block
// threadIdx.x == subhistogram index
static __global__ void mergeHistogramTex256Kernel(int *d_Result)
{
	__shared__ int data[BLOCK_N];

	//Reads are uncoalesced, but this final stage takes
	//only a fraction of total processing time
	data[threadIdx.x] = d_Result[threadIdx.x * BIN_COUNT + blockIdx.x];

	for (int stride = BLOCK_N / 2; stride > 0; stride >>= 1)
	{
		__syncthreads();

		if (threadIdx.x < stride)
			data[threadIdx.x] += data[threadIdx.x + stride];
	}

	if (threadIdx.x == 0)
		d_Result[blockIdx.x] = data[0];
}

////////////////////////////////////////////////////////////////////////////////
// Host interface to GPU histogram
////////////////////////////////////////////////////////////////////////////////

extern "C"
void checkCudaError()
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s.\n",
			hipGetErrorString(err));
		exit(2);
	}
}

//Maximum block count for histogram64kernel()
//Limits input data size to 756MB
//const int MAX_BLOCK_N = 16384;

//Internal memory allocation
//const int BLOCK_N2 = 32;

std::string GenerateHistogram(ID3D11Texture2D* dxTexture, int width, int height, int* Histogram)
{

	static hipGraphicsResource *cudaResource = nullptr;
	static ID3D11Texture2D* cudaTexture = nullptr;
	static int *d_histogram = nullptr;
	size_t HistogramSize = BIN_COUNT * sizeof(int);

	if (!cudaResource || cudaTexture != dxTexture)
	{
		cudaGraphicsD3D11RegisterResource(&cudaResource, dxTexture,
			hipGraphicsRegisterFlagsNone);

		cudaTexture = dxTexture;
		hipMalloc(&d_histogram, HistogramSize * 64);
	}

	hipGraphicsMapResources(1, &cudaResource);

	hipArray *cuArray = nullptr;
	hipGraphicsSubResourceGetMappedArray(&cuArray, cudaResource, 0, 0);

	hipBindTextureToArray(colorTex, cuArray);	

	histogramTex256Kernel << <BLOCK_N, THREAD_N >> > (d_histogram, width, height, width *height / 4);
	checkCudaError();

	mergeHistogramTex256Kernel << <BIN_COUNT, BLOCK_N >> > (d_histogram);
	checkCudaError();

	hipUnbindTexture(colorTex);
	checkCudaError();

	hipMemcpy(Histogram, d_histogram, HistogramSize, hipMemcpyDeviceToHost);
	//hipFree(d_histogram);

	hipGraphicsUnmapResources(1, &cudaResource);

	//hipUnbindTexture(ColorBuffer);

	////Allocate device and host memory for histogram
	//int *d_histogram[1];
	//size_t HistogramSize = 256 * sizeof(int);
	//hipMalloc(&d_histogram[0], HistogramSize);

	//int num_levels[1] = { 256 };
	//int lower_level[1] = { 0 };        // lower sample value boundary of lowest bin)
	//int upper_level[1] = { 255 };    // (upper sample value boundary of upper bin)

	//int num_row_samples = width;
	//int num_rows = height;
	//int num_pixels = width * height;
	//size_t row_stride_bytes = num_row_samples * sizeof(unsigned char) * 4;


	//// Determine temporary device storage requirements
	//void *d_temp_storage = NULL;
	//size_t temp_storage_bytes = 0;
	//hipcub::DeviceHistogram::MultiHistogramEven<4, 1>(d_temp_storage, temp_storage_bytes,
	//	cudaTexture, d_histogram, num_levels, lower_level, upper_level,
	//	num_pixels);// num_row_samples, num_rows, row_stride_bytes);


	//				// Allocate temporary storage
	//hipMalloc(&d_temp_storage, temp_storage_bytes);

	//// Compute histograms
	//hipcub::DeviceHistogram::MultiHistogramEven<4, 1>(d_temp_storage, temp_storage_bytes,
	//	cudaTexture, d_histogram, num_levels, lower_level, upper_level,
	//	num_pixels);//num_row_samples, num_rows, row_stride_bytes);

	//hipMemcpy(Histogram, d_histogram[0], HistogramSize, hipMemcpyDeviceToHost);

	//// Unmap and unregister the graphics resource
	//hipGraphicsUnmapResources(1, &cudaResource);
	//hipGraphicsUnregisterResource(cudaResource);

	//hipFree(d_temp_storage);
	//hipFree(d_histogram[0]);

	return {};
}